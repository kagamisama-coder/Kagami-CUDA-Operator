#include "utils.cuh"

int main() {
  const int M = 6144;
  const int N = 6144;
  const int K = 6144;
  CudaDeviceInfo();
  float *h_A = nullptr, *h_B = nullptr, *h_C = nullptr, *h_C_ref = nullptr;
  float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr, *d_C_ref = nullptr;

  int size_A = M * K * sizeof(float);
  int size_B = K * N * sizeof(float);
  int size_C = M * N * sizeof(float);

  h_A = (float *)malloc(size_A);
  h_B = (float *)malloc(size_B);
  h_C = (float *)malloc(size_C);
  h_C_ref = (float *)malloc(size_C);

  hipblasHandle_t handle;
  if (hipblasCreate(&handle)) {
    printf("Create cublas handle error.\n");
    exit(EXIT_FAILURE);
  };

  CUDA_CHECK(hipMalloc((void **)&d_A, size_A));
  CUDA_CHECK(hipMalloc((void **)&d_B, size_B));
  CUDA_CHECK(hipMalloc((void **)&d_C, size_C));
  CUDA_CHECK(hipMalloc((void **)&d_C_ref, size_C));

  // warmup
  std::cout << "============Warm up===========" << std::endl;
  init_matrix(h_A, M, K);
  init_matrix(h_B, K, N);
  CUDA_CHECK(hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice));
  test_kernel(0, d_A, d_B, d_C, M, N, K, handle);
  hipDeviceSynchronize();

  const int repeat_time = 10;

  for (int epoch = 0; epoch < repeat_time; epoch++) {
    std::cout << "============Epoch " << epoch + 1 << "===========" << std::endl;

    init_matrix(h_A, M, K);
    init_matrix(h_B, K, N);

    CUDA_CHECK(hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice));

    test_kernel(0, d_A, d_B, d_C_ref, M, N, K, handle);

    hipMemcpy(h_C_ref, d_C_ref, size_C, hipMemcpyDeviceToHost);

    test_kernel(3, d_A, d_B, d_C, M, N, K, handle);

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    verify_matrix(h_C, h_C_ref, M, N);
  }

  free(h_A);
  free(h_B);
  free(h_C);
  free(h_C_ref);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_C_ref);

  return 0;
}